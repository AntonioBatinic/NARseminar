#include "hip/hip_runtime.h"
#include "filtering.h"

#include <>
#include <hip/hip_runtime.h>

#include <cuda/cuda_utilities.h>
#include <cuda/uint_util.hcu>
#include <cuda/float_util.hcu>

#include <common/error.h>

#include <math.h>

#define PI 3.14159265

//Textures
texture<float, 1> tex;
texture<float, 1> tex_red;
texture<float, 1> tex_blue;
texture<float, 1> tex_green;


__host__ __device__
float gaussian1d(float x, float sigma)
{
	float variance = pow(sigma,2);
	float exponent = -pow(x,2)/(2*variance);
	return expf(exponent) / sqrt(2 * PI * variance);
}

inline __device__
float gaussian1d_gpu(float x, float sigma)
{
	float variance = __powf(sigma,2);
	float power = pow(x,2);
	float exponent = -power/(2*variance);
	return __expf(exponent) / sqrt(2 * PI * variance);
}

inline __device__
float gaussian1d_gpu_reg(float x, float variance, float sqrt_pi_variance)
{
	float gaussian1d = -(x*x)/(2*variance);
	gaussian1d = __expf(gaussian1d);
	gaussian1d /= sqrt_pi_variance;
	return gaussian1d;
}

__host__ __device__
float gaussian2d(float x, float y, float sigma)
{
	float variance = pow(sigma,2);
	float exponent = -(pow(x,2) + pow(y,2))/(2*variance);
	return expf(exponent) / (2 * PI * variance);
}


float* generateGaussianKernel(int radius, float sigma)
{
	int area = (2*radius+1)*(2*radius+1);
	float* res = new float[area];

	for(int x = -radius; x <= radius; x++)
		for(int y = -radius; y <= radius; y++)
		{
			
			int position = (x+radius)*(radius*2+1) + y+radius; 
			res[position] = gaussian2d(x,y,sigma);
		}
	return res;
}



__global__
void bilateralFilterGPU_v(float* output, uint2 dims, int radius, float* kernel, float variance, float sqrt_sigma)
{
	const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

	uint2 pos = idx_to_co(idx,dims);

	if(pos.x >= dims.x || pos.y >= dims.y) return;

	float currentColor = tex1Dfetch(tex, idx);

	float res = 0.0f;
	float normalization = 0.0f;

	for(int i = -radius; i <= radius; i++) {
		for(int j = -radius; j <= radius; j++) {
			
			int x_sample = pos.x+i;
			int y_sample = pos.y+j;

			
			if( x_sample < 0) x_sample = -x_sample;
			if( y_sample < 0) y_sample = -y_sample;
			if( x_sample > dims.x - 1) x_sample = dims.x - 1 - i;
			if( y_sample > dims.y - 1) y_sample = dims.y - 1 - j;

			float tmpColor = tex1Dfetch(tex, y_sample*dims.x + x_sample);
			
			float gaussian1d = -((currentColor - tmpColor)*(currentColor - tmpColor))/(2*variance);
			gaussian1d = __expf(gaussian1d);
			gaussian1d /= sqrt_sigma;
			float gauss_spatial = kernel[co_to_idx(make_uint2(i+radius,j+radius),make_uint2(radius*2+1,radius*2+1))];
			
			float weight = gauss_spatial * gaussian1d;

			normalization = normalization + weight;
			res = res + (tmpColor * weight);

		}
	}
	
	output[idx] = res / normalization;
}


/*
The gaussian kernel for the spatial domain is precalculated before the launch
The gaussian for the range is using the hardware implementations
The input data is cached through a 3 x 1D textures, output is 3 different areas, for r g and b individually
*/
void bilateralFiltering_v(RGB* data, int width, int height,int radius, float sigma_spatial, float sigma_range)
{
	unsigned int numElements = width * height;

    //Input data
    float* d_red;
	float* d_blue;
	float* d_green;
    hipMalloc( (void**) &d_red, numElements*sizeof(float));
	hipMalloc( (void**) &d_blue, numElements*sizeof(float));
	hipMalloc( (void**) &d_green, numElements*sizeof(float));

	//Output data
    float* d_red_out;
	float* d_blue_out;
	float* d_green_out;
    hipMalloc( (void**) &d_red_out, numElements*sizeof(float));
	hipMalloc( (void**) &d_blue_out, numElements*sizeof(float));
	hipMalloc( (void**) &d_green_out, numElements*sizeof(float));

	float* red = new float[numElements];
	float* green = new float[numElements];
	float* blue = new float[numElements];

	for(int i = 0; i < numElements; i++)
	{
		red[i] = data[i].R;
		green[i] = data[i].G;
		blue[i] = data[i].B;
	}

	//Copy image to device
    hipMemcpy(	d_red, red, numElements*sizeof(float), hipMemcpyHostToDevice );

	//Set up kernel
	float* kernel = generateGaussianKernel(radius,sigma_spatial);
	float* d_Kernel;
	hipMalloc( (void**) &d_Kernel, (2*radius+1)*(2*radius+1) * sizeof(float));
	hipMemcpy( d_Kernel, kernel, (2*radius+1)*(2*radius+1)* sizeof(float), hipMemcpyHostToDevice);

    // setup dimensions of grid/blocks.
    dim3 blockDim(192,1,1);
    dim3 gridDim((unsigned int) ceil((double)(numElements/blockDim.x)), 1, 1 );

	const hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture(0, &tex, d_red, &desc, numElements*sizeof(float));

    // invoke kernel
	bilateralFilterGPU_v<<< gridDim, blockDim >>>
		( d_red_out, make_uint2(width,height), radius, d_Kernel, sigma_range*sigma_range,sqrt(2*PI*sigma_range*sigma_range)); //sqrt(2 * PI * variance)

	hipMemcpy( d_green, green, numElements*sizeof(float), hipMemcpyHostToDevice );
	
	//Wait before starting the next
	hipDeviceSynchronize();
	
	hipUnbindTexture(tex);
	hipBindTexture(0, &tex, d_green, &desc, numElements*sizeof(float));

    // invoke kernel
	bilateralFilterGPU_v<<< gridDim, blockDim >>>
		( d_green_out, make_uint2(width,height), radius, d_Kernel, sigma_range*sigma_range,sqrt(2*PI*sigma_range*sigma_range));

	hipMemcpy( d_blue, blue, numElements*sizeof(float), hipMemcpyHostToDevice );
	//Wait before starting the next
	hipDeviceSynchronize();
	
	hipUnbindTexture(tex);
	hipBindTexture(0, &tex, d_blue, &desc, numElements*sizeof(float));

    // invoke kernel
	bilateralFilterGPU_v<<< gridDim, blockDim >>>
		( d_blue_out, make_uint2(width,height), radius, d_Kernel, sigma_range*sigma_range,sqrt(2*PI*sigma_range*sigma_range));

	hipDeviceSynchronize(); //sync before freeing
	hipUnbindTexture(tex);

    // copy data to host
    hipMemcpy(	red, d_red_out, numElements*sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy(	green, d_green_out, numElements*sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy(	blue, d_blue_out, numElements*sizeof(float), hipMemcpyDeviceToHost );

	hipFree(d_red);
	hipFree(d_green);
	hipFree(d_blue);
	hipFree(d_red_out);
	hipFree(d_green_out);
	hipFree(d_blue_out);

	for(int i = 0; i < numElements; i++)
	{
		data[i].R = red[i];
		data[i].G = green[i];
		data[i].B = blue[i];
	}

	//Error handling
    CHECK_FOR_CUDA_ERROR();
}


